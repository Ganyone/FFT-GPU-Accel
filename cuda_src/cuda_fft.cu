#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <hip/hip_runtime.h>
#include <cmath>
#include <algorithm>
#include <stdio.h>
#include <iomanip>
#include "common.h"
#include "cmdline.h"
#include "hip/hip_runtime_api.h"

using namespace std;
//typedef float Type;
typedef double Type;
//const Type pi=3.1415926535897932384626433832795028;
const Type pi=3.141592653589793238462643383279502884197169399375105820974944;

__global__ void fft(Type* devI_r,Type* devI_i,Type* devW_r,Type* devW_i,int nElem, int l, int s, int m){
	const int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int wid;
	int p,q;
	Type Tp_r,Tp_i,Tq_r,Tq_i,Wr,Wi,TqW_r,TqW_i;
	wid = tid + s*nElem/2;
	p = 2*wid-wid % m-nElem*s;
	q = p + m;
	Wr = devW_r[wid];
	Wi = devW_i[wid];
	Tp_r = devI_r[p];
	Tp_i = devI_i[p];
	Tq_r = devI_r[q];
	Tq_i = devI_i[q];
	TqW_r = Tq_r*Wr - Tq_i*Wi;
	TqW_i = Tq_r*Wi + Tq_i*Wr;
	devI_r[p] = Tp_r + TqW_r;
	devI_i[p] = Tp_i + TqW_i;

	devI_r[q] = Tp_r - TqW_r;
	devI_i[q] = Tp_i - TqW_i;
}

void bit_reverse(Type *A,int sz){
    for(int i=0,j=0;i<sz-1;++i){
        if(i<j) {swap(A[i],A[j]);}
         int k=sz/2;
        while(k<=j){
            j-=k;
            k/=2;
        }
        j+=k;
    }
}

void w_fun(Type *W_r, Type *W_i, int nElem){
	int s=0;
    for(int m=2; m<=nElem; m*=2){
    	int i=0;
        for(int k=0; k<nElem; k+=m){
            for(int j=0;2*j<m;j++){
                W_r[s*nElem/2+i] =  cos(2*pi*(k+j)/m);
                W_i[s*nElem/2+i] = -sin(2*pi*(k+j)/m);
                i++;
            }
        }
        s++;
    }
}

void InitialData(Type *I_r,Type *I_i,int nElem,string data_file){
	FILE* f;
	const char* file_name = data_file.c_str();
	f = fopen(file_name,"r+");
	int i=0;
	while(i<nElem && fscanf(f,"%lf",I_r+i)==1){
		++i;
		I_i[i] = 0;
	}
//    for(int i=0;i<nElem;i++){
//    	I_r[i] = i+1;
//    	I_i[i] = 0;
//    };
    bit_reverse(I_r,nElem);
    bit_reverse(I_i,nElem);
}
int main(int argc, char *argv[]){
	cmdline::parser a;
	a.add<string>("data", 'd', "data file", true, "");
	a.add<int>("num", 'n', "data number", false, 20, cmdline::range(2, 23));
	a.add<string>("result", 'r', "result", false, "./fftresult.txt");

	a.parse_check(argc, argv);
	int N,l;
//	scanf("%d",&l);
	l = a.get<int>("num");
	string data_file = a.get<string>("data");
	string result_file = a.get<string>("result");
	N = 1<<l;
	hipProfilerStart();
	Type *W_r,*W_i,*I_r,*I_i;
	unsigned int flags = hipHostMallocMapped;
	CHECK( hipHostAlloc( (Type**)&W_r,sizeof(Type)*l*N/2+100,flags) );
	CHECK( hipHostAlloc( (Type**)&W_i,sizeof(Type)*l*N/2+100,flags) );
	CHECK( hipHostAlloc( (Type**)&I_r,sizeof(Type)*N+100,flags) );
	CHECK( hipHostAlloc( (Type**)&I_i,sizeof(Type)*N+100,flags) );
    w_fun(W_r,W_i,N);
    InitialData(I_r,I_i,N,data_file);
    Type *devI_r, *devI_i;


    CHECK( hipMalloc( (Type**)&devI_r, sizeof(Type)*N+100 ) );
    CHECK( hipMalloc( (Type**)&devI_i, sizeof(Type)*N+100 ) );
    Type *devW_r, *devW_i;
    CHECK( hipMalloc( (Type**)&devW_r, sizeof(Type)*l*N/2+100 ) );
    CHECK( hipMalloc( (Type**)&devW_i, sizeof(Type)*l*N/2+100 ) );
    CHECK( hipMemcpy(devI_r, I_r, sizeof(Type)*N+100, hipMemcpyHostToDevice) );
    CHECK( hipMemcpy(devI_i, I_i, sizeof(Type)*N+100, hipMemcpyHostToDevice) );
    CHECK( hipMemcpy(devW_r, W_r, sizeof(Type)*l*N/2+100, hipMemcpyHostToDevice) );
    CHECK( hipMemcpy(devW_i, W_i, sizeof(Type)*l*N/2+100, hipMemcpyHostToDevice) );
    hipHostFree(W_r);
    hipHostFree(W_i);
    int m=1;
    hipEvent_t start, stop;
	CHECK( hipEventCreate(&start) );
	CHECK( hipEventCreate(&stop) );
	CHECK( hipEventRecord(start, 0) );
	int iStart,iEnd;
	double Time = 0;
	iStart = clock();
//	double iStart = cpuSecond();
    for (int s=0;s<l;s++){
        if(N<=1024)
        	fft<<<1,N/2>>>(devI_r,devI_i,devW_r,devW_i,N,l,s,m);
        else
        	fft<<<N/1024,512>>>(devI_r,devI_i,devW_r,devW_i,N,l,s,m);
        m*=2;
    }
    hipDeviceSynchronize();  // 同步函数
//    double iElaps = cpuSecond() - iStart;
    iEnd = clock();
    Time = (iEnd - iStart);
    Time=(Time+0.0)/(CLOCKS_PER_SEC);
    double onceTime = Time/N;
    int p1 = data_file.find_last_of('/');
    string dir_loc = data_file.substr(0,p1);
    string r_file = dir_loc + "/fft_accel_tmp";
    const char * target_file = r_file.c_str();
    FILE *f;
    f = fopen(target_file,"w+");
    fprintf(f,"%f\t%d\t%.12f\n",Time,N,onceTime);
    fclose(f);
	CHECK( hipEventRecord(stop, 0) );
	CHECK( hipEventSynchronize(stop) );
	CHECK( hipEventSynchronize(stop) );
	float elapsedTime;
	CHECK( hipEventElapsedTime(&elapsedTime,start,stop) );
    hipFree(devW_r);
    hipFree(devW_i);
    CHECK( hipMemcpy(I_r, devI_r, sizeof(Type)*N+100, hipMemcpyDeviceToHost) );
    CHECK( hipMemcpy(I_i, devI_i, sizeof(Type)*N+100, hipMemcpyDeviceToHost) );
    ofstream outFile(result_file);

	for(int i=0;i<N;i++){
		outFile<<setprecision(10)<<I_r[i]<<" + "<<I_i[i]<<"i\n";
	}
	printf( "time : %3.1fms\n",elapsedTime);
	CHECK( hipEventDestroy(start) );
	CHECK( hipEventDestroy(stop) );
    hipFree(devI_r);
    hipFree(devI_i);
    hipHostFree(I_r);
    hipHostFree(I_i);
    hipProfilerStop();
    hipDeviceReset();
    return 0;
}
